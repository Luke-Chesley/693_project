#include "llmc/cuda_common.h"
#include <cstdio>
#define TESTING

#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "train_gpt2.cu"

void common_start_local(bool override_enable_tf32 = true, bool print_device_info = true) {

    // get CUDA device infos
    cudaCheck(hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx));
    if (print_device_info) {
        printf("[System]\n");
        printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;

    //printf("ENABLE_tf32:%d\n",enable_tf32);

    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    #ifdef ENABLE_CUDNN
    create_cudnn();
    #endif
}
void gpt2_init_common_local(GPT2 *model) {
    // common inits outside of the model weights
    // memory lazily initialized in forward()
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->accumulated_mean_loss = NULL;
    model->cpu_losses = NULL;
    // the B,T params are determined and set, fixed on first batch in forward()
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss, set at end of forward()
    model->params_memory = NULL;
    // memory lazily initialized in backward()
    model->grads_memory = NULL;
    model->workload_indices = NULL; // on cpu, for encoder_backward
    model->bucket_info = NULL; // on cpu, for encoder_backward
    // memory lazily initialized in update()
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    // other default settings
    model->rng_state = 13371337 + multi_gpu_config.process_rank; // used in stochastic rounding
    model->use_master_weights = 1; // safe default: do keep master weights in fp32
    model->init_state = true;
    model->recompute = 1; // good default: recompute gelu but not layernorm
    model->gelu_fusion = 0; //deviceProp.major >= 9 ? 2 : 0; // default: off for now (default must match main())
}


void gpt2_build_from_checkpoint_local(GPT2 *model, const char* checkpoint_path, bool weight_init=true) {
    // If weight_init is true, we will load the weights from this checkpoint .bin file
    // We sometimes want this to be false, if we are going to initialize these weights from
    // the master weights that are instead stored in the state .bin file.
    // In that case, this function mostly loads the model hyperparameters from the header.

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }

    // check if the precision mode of the checkpoing matches the model precision
    if (weight_init) {
        if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
            fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
            exit(EXIT_FAILURE);
        }
        if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
            fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_gpt2cu PRECISION=FP32`\n");
            fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
            exit(EXIT_FAILURE);
        }
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate memory for the model parameters
    gpt2_allocate_weights(model);

    // read in the parameters if weight_init is true
    if (weight_init) {
        assert(model->params_memory != NULL);
        file_to_device(model->params_memory, model_file, model->num_parameters_bytes, IO_BUF_SIZE, main_stream);
    }
    fcloseCheck(model_file);

    // only return from this function once we are certain the params are ready on the GPU
    cudaCheck(hipDeviceSynchronize());
}

// propagate inputs through the network to produce logits.
// right now, this function is fully synchronous with the host
void gpt2_forward_local(GPT2 *model, const int* inputs, size_t B, size_t T) {
    NVTX_RANGE_FN();
    // we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    // validate B,T are not larger than the values used at initialisation
    // (smaller B,T are okay for inference only)
    if (B > model->batch_size || T > model->seq_len) {
        printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
        exit(EXIT_FAILURE);
    }

    // copy inputs/targets to the model
    // inputs are copiede to model-> inputs on the device
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    // validate inputs, all indices must be in the range [0, V)
    // we can do this while the copies are already underway
    tokenCheck(inputs, B*T, V);

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C, main_stream); // encoding goes into residual[0]

    // first layernorm isn't fused
    layernorm_forward((model->recompute < 2) ? acts.ln1 : acts.lnf, acts.ln1_mean, acts.ln1_rstd, acts.encoded, params.ln1w, params.ln1b, B, T, C, main_stream);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;
        floatX* scratch = (floatX*)acts.output; // used for non-cudnn attention, fcproj, attproj, etc.

        // now do the forward pass
        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        if (T != model->seq_len) { // unused parts of attention buffer must be zeroed (T-dependent)
            cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX)));
        }
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH, main_stream);
        #endif

        matmul_forward_cublaslt(scratch, l_atty, l_attprojw, l_attprojb, B, T, C, C, main_stream);
        fused_residual_forward5(l_residual2, l_ln2, l_ln2_mean, l_ln2_rstd, residual, scratch, l_ln2w, l_ln2b, B*T, C, main_stream);
        matmul_forward_cublaslt(l_fch_gelu, l_ln2, l_fcw, l_fcb, B, T, C, 4*C, main_stream, l_fch, model->gelu_fusion);
        matmul_forward_cublaslt(scratch, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C, main_stream);
        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + (l + 1) * B * T * C : acts.lnf;
            float* l_ln1_mean = acts.ln1_mean + (l + 1) * B * T;
            float* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            const floatX* l_ln1b = params.ln1b + (l + 1) * C;
            fused_residual_forward5(l_residual3, l_ln1, l_ln1_mean, l_ln1_rstd, l_residual2, scratch, l_ln1w, l_ln1b,
                                    B * T, C, main_stream);
        } else {
            fused_residual_forward5(l_residual3, acts.lnf, acts.lnf_mean, acts.lnf_rstd, l_residual2, scratch,
                                    params.lnfw, params.lnfb,
                                    B * T, C, main_stream);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    cudaCheck(hipDeviceSynchronize());
}





int main(){
    char nccl_init_method[256] = "mpi"; 
    char server_ip[256] = "";  // doesn't matter when using MPI
    char fs_path[256] = "";    // doesn't matter when using MPI

    // Common initialization
    common_start_local(true, true); // Enable TF32 and print device info


    printf("Precision mode: %i\n",PRECISION_MODE);

    // MODEL
    GPT2 model;
    gpt2_init_common_local(&model);

    if (PRECISION_MODE == 0){
        gpt2_build_from_checkpoint_local(&model, "gpt2_124M.bin");

    } else if (PRECISION_MODE == 2) {
        gpt2_build_from_checkpoint_local(&model, "gpt2_124M_bf16.bin");
    }

    // Initialize multi_gpu_config after model is built
    multi_gpu_config = multi_gpu_config_init(1, 0, 1, server_ip, fs_path, nccl_init_method); // Adjust parameters as needed

    // Debugging: Print model.num_parameters and shard_num_parameters
    printf("Model Parameters: %lu\n", model.num_parameters);
    printf("Shard Num Parameters: %zu\n", multi_gpu_config.shard_num_parameters);

    int B = 1;
    int T = 1024;
    int genT = 100;

    // Allocate memory for inference
    gpt2_allocate_state(&model, B, T); // B=1, T=1024

    printf("batch size: %d\n", B);
    printf("sequence length: %d\n", T);

    // TOKENIZER
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));
    unsigned long long sample_rng_state = (unsigned long long)time(NULL);
    int eot_token = tokenizer.eot_token;

    for (int i = 0; i < B * T; i++){
      gen_tokens[i] = eot_token;
    }


    TokenArray query_tokens = read_tokens("encoded_temp.bin");

    for (uint32_t i = 0; i < query_tokens.num_tokens;i++){
        gen_tokens[i] = query_tokens.token_ids[i];
    }

    


    hipEvent_t start, stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));

    // Record the start event
    cudaCheck(hipEventRecord(start, 0));


    printf("generating:\n---\n");
    for (int t = 0; t < genT; t++){
        if (t < query_tokens.num_tokens){
            const char* token_str = tokenizer_decode(&tokenizer, gen_tokens[t]);
            printf("%s",token_str);

        } else{
        gpt2_forward_local(&model, gen_tokens, B, T);

        floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
        cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));

        for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }

        float coin = random_f32(&sample_rng_state);
        int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
        gen_tokens[t] = next_token;

        // if (next_token == eot_token){
        //     break;
        //     }

        if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                    //printf(" %i\n",next_token);
                } else {
                    // fall back to printing the token id
                    printf("%d", next_token);
                }
                fflush(stdout);


        }   
    }
    printf("\n---\n");
    

    // Record the stop event
    cudaCheck(hipEventRecord(stop, 0));

    // Wait for the stop event to complete
    cudaCheck(hipEventSynchronize(stop));

    // Calculate elapsed time
    float milliseconds = 0;
    cudaCheck(hipEventElapsedTime(&milliseconds, start, stop));

    // Destroy CUDA events
    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));

    // Calculate average time per token
    double average_time_per_token = milliseconds / (double)(genT - 1); // genT-1 tokens generated

    // Print the results
    printf("Total generation time: %.3f ms\n", milliseconds);
    printf("Average time per token: %.3f ms\n", average_time_per_token);

    // Clean up resources
    tokenizer_free(&tokenizer);
    free(query_tokens.token_ids);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    multi_gpu_config_free(&multi_gpu_config);
    gpt2_free(&model);
    common_free(model);


}
